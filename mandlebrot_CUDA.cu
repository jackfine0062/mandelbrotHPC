#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdbool.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <inttypes.h>
#include <unistd.h>

#include "matrix.h"
#include "util.h"

#include <hip/hip_runtime.h>

#define CHECK(call)                                                       \
{                                                                         \
   const hipError_t error = call;                                        \
   if (error != hipSuccess)                                              \
   {                                                                      \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
      exit(1);                                                            \
   }                                                                      \
}

// INSTRUCTIONS
// Compile take 4 arguements
// iterations
// width
// height
// Set Num (0 = Julia, 1 = Mandlebrot)
// gcc -Wall -O3 -march=native -c matrix.c util.c
// nvcc -O3 *.cu *.o -o mandlebrot_CUDA -lm
// ./mandlebrot_CUDA 1000 1000 1000 0

// Plot after you compile and run
// python3 plot.py

__global__ void julia_cuda(float* pixels, size_t width, size_t height, int max_iteration) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    int R = 20;  // choose R > 0 such that R**2 - R >= sqrt(cx**2 + cy**2)
    float cx = -0.7269;
    float cy = 0.1889;

    if (x < width && y < height) {

        float scaled_x = (1.f - (-2.5f)) / width * x - 2.5f;
        float scaled_y = (1.f - (-1.f)) / height * y - 1.f;
        int iteration = 0;
        while (scaled_x*scaled_x + scaled_y*scaled_y <= R*R && iteration < max_iteration) {
            float xtemp = scaled_x*scaled_x - scaled_y*scaled_y;
            scaled_y = 2*scaled_x*scaled_y + cy;
            scaled_x = xtemp + cx;
            iteration += 1;
        }
        pixels[y*width + x] = iteration;
    }
}

__global__ void mandle_cuda(float* pixels, size_t width, size_t height, int max_iteration) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {

        float scaled_x = (1.f - (-2.5f)) / width * x - 2.5f;
        float scaled_y = (1.f - (-1.f)) / height * y - 1.f;
        float real = 0.f;
        float imaginary = 0.f;
        int iteration = 0;
        while (real*real + imaginary*imaginary <= 2*2 && iteration <= max_iteration) {
            float xtemp = real*real - imaginary*imaginary + scaled_x;
            imaginary = 2*real*imaginary + scaled_y;
            real = xtemp;
            iteration += 1;
        }
        pixels[y*width + x] = iteration;
    }
}

void julia_set(Matrix* pixels, int max_iteration) {

    size_t width = pixels->cols;
    size_t height = pixels->rows;

    float *d_a;
    size_t pixel_bytes = pixels->size*sizeof(float);
    CHECK(hipMalloc(&d_a, pixel_bytes));

    int dimx = 4, dimy = 192;
    dim3 block(dimx, dimy);
    dim3 grid((width + dimx - 1) / dimx, (height + dimy - 1) / dimy);

    julia_cuda<<<grid, block>>>(d_a, width, height, max_iteration);

    CHECK(hipMemcpy(pixels->data, d_a, pixel_bytes, hipMemcpyDeviceToHost));
    CHECK(hipFree(d_a));
}

void mandelbrot(Matrix* pixels, int max_iteration) {

    size_t width = pixels->cols;
    size_t height = pixels->rows;

    float *d_a;
    size_t pixel_bytes = pixels->size*sizeof(float);
    CHECK(hipMalloc(&d_a, pixel_bytes));

    int dimx = 4, dimy = 192;
    dim3 block(dimx, dimy);
    dim3 grid((width + dimx - 1) / dimx, (height + dimy - 1) / dimy);

    mandle_cuda<<<grid, block>>>(d_a, width, height, max_iteration);

    CHECK(hipMemcpy(pixels->data, d_a, pixel_bytes, hipMemcpyDeviceToHost));
    CHECK(hipFree(d_a));
}

void multibrot(Matrix* pixels, int max_iteration) {

    size_t width = pixels->cols;
    size_t height = pixels->rows;

    for (int x = 0; x < width; x++) {
        for (int y = 0; y < height; y++) {

            // printf("%d, %d\n", x, y);
            float scaled_x = (1.f - (-2.5f)) / width * x - 2.5f;
            float scaled_y = (1.f - (-1.f)) / height * y - 1.f;
        
            int iteration = 0;
            while (scaled_x*scaled_x + scaled_y*scaled_y <= (2*2) && iteration < max_iteration) {

                int a = scaled_x;
                int b = scaled_y;

                float xtmp= (scaled_x * scaled_x * scaled_x * scaled_x * scaled_x) -
                            10 * (scaled_x * scaled_x * scaled_x) * (scaled_y * scaled_y) + 5 * scaled_x * 
                            (scaled_y * scaled_y * scaled_y * scaled_y) + a;

                scaled_y = 5 * (scaled_x * scaled_x * scaled_x * scaled_x)*scaled_y-
                10 * (scaled_x * scaled_x) * (scaled_y * scaled_y * scaled_y) + (scaled_y * scaled_y * scaled_y * scaled_y * scaled_y) + b;
                scaled_x = xtmp;

                iteration = iteration + 1;
            }

            pixels->data[y*pixels->rows + x] = iteration;

            // printf("%d\n", iteration);
            // printf("%d, %d\n", x, y);
        
            // if (iteration = max_iteration)
            //     colour = black;
            // else
            //     colour = iteration;
        
            // plot(scaled_x, scaled_y, colour)
        }
    }

}

int main(int argn, const char* argv[])
{

    // default command-line options
    int iterations = 100;   
    int width = 100; 
    int height = 100; 
    // Specify set to plot (0 Julia, 1 Mandelbrot, 2 Multibrot)
    int sets = 1;

    if (argn >= 2) {        
        int n = atoi(argv[1]);        
        if (n < 1) {
            printf("Iterations must be positive integer\n"); 
            return 1; 
        }        
        iterations = n;
    }
    if (argn >= 3) {        
        int n = atoi(argv[2]);        
        if (n < 1) {
            printf("Width must be positive integer\n"); 
            return 1; 
        }        
        width = n;
    }
    if (argn >= 4) {        
        int n = atoi(argv[3]);        
        if (n < 1) {
            printf("Height must be positive integer\n"); 
            return 1; 
        }
        height = n;
    }
    if (argn >= 5) {        
        int n = atoi(argv[4]);        
        if (n < 0 || n > 2) {
            printf("Specify 0 for Julia set, 1 for Mandlebrot set, 2 for Multibrot \n"); 
            return 1; 
        }
        sets = n;
    }
    // seed random number generator
    srand(time(NULL));

    // start the timer
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    Matrix* pixels = matrix_zeros(width, height);

    if(sets == 0) {
        julia_set(pixels, iterations);
    } else if(sets == 1) {
        mandelbrot(pixels, iterations);
    } else if(sets == 2) {
        multibrot(pixels, iterations);
    }
    matrix_to_npy_path("pixels.npy", pixels);
    // get the end and computation time
    clock_gettime(CLOCK_MONOTONIC, &end);
    double time = get_time_diff(&start, &end);
    printf("%f secs\n", time);

    matrix_free(pixels);

	return 0;
}
